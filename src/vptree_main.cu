#include <iostream>
#include <random>
#include <chrono>
#include "vptree.cuh"
using namespace std;

#define MIN_VALUE               -1000.0
#define MAX_VALUE               1000.0

#define NUM_OF_INPUTS           2
#define INPUT_ERROR             -1 
#define MEMORY_ERROR            -2

void cudaGPUDetails();
void checkGpuMem();
void initDataset(float **dataset, unsigned long n, unsigned long d);



int main(int argc, char *argv[]){
    unsigned long n, d;
    float *h_dataset;

    // read inputs
    if(argc != (NUM_OF_INPUTS + 1)){
        cout << "Wrong number of inputs. Exiting..." <<endl;
        exit(-1);
    }else{
        n = atoi(argv[1]);
        d = atoi(argv[2]);
    }

    cudaGPUDetails();
    checkGpuMem();

    // initialize dataset
    initDataset(&h_dataset, n, d);

    // create tree
    cout << "Building tree";
    auto start = std::chrono::high_resolution_clock::now();
    createVPTree(h_dataset, n, d);
    auto finish = std::chrono::high_resolution_clock::now();
    auto gpu_time = std::chrono::duration_cast<std::chrono::nanoseconds>(finish-start).count();
    cout << "   Finished in: " << (float)gpu_time/(10e3) << " us" << endl; 


    // free resources
    free(h_dataset);     
    
    return 0;
}


void initDataset(float **dataset, unsigned long n, unsigned long d){
    *dataset = (float *)malloc(n * d * sizeof(float));

    cout << "Initialazing dataset of " << n << " points of " << d << " dimensions" << endl;
    
    if(*dataset == NULL){
        cout << "Not enough memory. Aborting ..." << endl;
        free(*dataset);
        exit(MEMORY_ERROR);
    }else{
        // initialize random generator engine
        unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
        default_random_engine generator(seed);
        uniform_real_distribution<float> distribution(MIN_VALUE, MAX_VALUE);

        unsigned long length = n * d;

        for(unsigned long i = 0; i < length; i++){
            (*dataset)[i] = distribution(generator);
        }
    }

    cout << "Dataset initialized succesfully" << endl;
}


void cudaGPUDetails(){
    int n_devices;
    hipGetDeviceCount(&n_devices);

    cout << "******** Getting GPU information ********" << endl;

    for (int dev = 0; dev < n_devices; dev++) {
        hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, dev);

        if(n_devices == 0){
            if(device_prop.major == 9999 && device_prop.minor == 9999){
                cout << "No Cuda GPU has been detected" << endl;
                exit(-1);
            }else if(n_devices == 1){
                cout << "Found 1 device supporting CUDA" << endl;
            }else{
                cout << "There are " << n_devices << " supporting CUDA" << endl;
            }
        }

        cout << "Device " << dev << " name: " << device_prop.name << endl;
        cout << "Computatial Capabilities " << device_prop.major << "." << device_prop.minor << endl;
        cout << "Maximum global memory size: " << device_prop.totalGlobalMem << endl;
        cout << "Maximum Constant memory size: " << device_prop.totalConstMem << endl;
        cout << "Maximun shared memory size per block: " << device_prop.sharedMemPerBlock << endl;
        cout << "Maximum block dimensions: " << device_prop.maxThreadsDim[0] << " x " << device_prop.maxThreadsDim[1] << " x " << device_prop.maxThreadsDim[2] << endl;
        cout << "Maximum grid dimensions: " << device_prop.maxGridSize[0] << " x " << device_prop.maxGridSize[1] << " x " << device_prop.maxGridSize[2] << endl;
        cout << "Warp size: " << device_prop.warpSize << endl;
    } 

    cout << "*****************************************" << endl << endl << endl;
}



void checkGpuMem(){
    float free_m,total_m,used_m;
    size_t free_t,total_t;

    hipMemGetInfo(&free_t,&total_t);

    free_m =(uint)free_t/1048576.0 ;
    total_m=(uint)total_t/1048576.0;
    used_m=total_m-free_m;

    cout << "Free mem is:" << free_m <<"MB from a total of: " << total_m << "MB while: "<< used_m <<" MB are already used!" << endl;
}
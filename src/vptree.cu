#include "hip/hip_runtime.h"
#include "vptree.cuh"


void createVPTree(float *dataset, int n, int d){
    float *d_dataset = NULL;
    float *d_distances = NULL;
    float *d_indexes = NULL;

    unsigned long length = n * d;

    // transfer dataset to device
    hipMalloc(&d_dataset, length * sizeof(float));         // keeps the dataset
    hipMalloc(&d_distances, n * sizeof(float));            // keeps the current distance results in each level
    hipMalloc(&d_indexes, n * sizeof(float));              // index map so we dont need to swap the whole element - only their indexes

    hipMemcpy(d_dataset, dataset, length * sizeof(float),  hipMemcpyHostToDevice);

    initIndexes(d_indexes, n);


    auto start = std::chrono::high_resolution_clock::now();
    parallelReduce(d_distances, d_dataset, d_indexes, n, d, 2);
    auto finish = std::chrono::high_resolution_clock::now();
    auto gpu_time = std::chrono::duration_cast<std::chrono::nanoseconds>(finish-start).count();
    cout << "Gpu: " << gpu_time << endl; 


    hipFree(d_dataset);
    hipFree(d_distances);
    hipFree(d_indexes);
}


void initIndexes(float *d, int n){
    dim3 block(512, 1);
    dim3 grid(CEIL_DIV(n, 512), 1);
    cudaInitIndexes<<<block, grid>>>(d, n);
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess){
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
        exit(SYNCH_CUDA_ERROR);
    }
		
	if (errAsync != hipSuccess){
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
        exit(ASYNC_CUDA_ERROR);
    }
}


__global__ void cudaInitIndexes(float *d, int n){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  
    if(thread_id < n){
        d[thread_id] = thread_id;
    }
}

#include "hip/hip_runtime.h"
#include "vptree.cuh"


void createVPTree(float *dataset, int n, int d){
    float *d_dataset = NULL;
    float *d_distances = NULL;
    float *d_indexes = NULL;
    float *d_tree = NULL;
    float *d_vp_points = NULL;

    unsigned long length = n * d;
    unsigned long tree_depth = log2(n);
    unsigned long tree_size = n * tree_depth;

    unsigned long vp_points_size = n - 1;



    // transfer dataset to device
    hipMalloc(&d_tree, tree_size * sizeof(float));
    hipMalloc(&d_dataset, length * sizeof(float));         // keeps the dataset
    hipMalloc(&d_distances, n * sizeof(float));            // keeps the current distance results in each level
    hipMalloc(&d_indexes, n * sizeof(float));              // index map so we dont need to swap the whole element - only their indexes
    hipMalloc(&d_vp_points, vp_points_size * sizeof(float));

    hipMemcpy(d_dataset, dataset, length * sizeof(float),  hipMemcpyHostToDevice);


    initIndexes(d_indexes, n);
    
    for(int i = 1; i <= n/2; i <<= 1){
        parallelDistance(d_distances, d_dataset, d_indexes, &d_vp_points[i-1], n, d, i);    // calculate distances
        copyIndexes(&d_tree[(unsigned int)log2(i) * n], d_indexes, n);                      // store indexes in tree level
        bitonic(d_distances, d_indexes, n, DESCENDING, i);                                  // sorted in descending order so as to suffle the vantage point 
    }
    
    
    #ifdef GLOBAL_SYNCHRONIZATION
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();



    if (errSync != hipSuccess)
        cout << "Sync kernel error: " << hipGetErrorString(errSync) << " in vp tree" << endl;
	if (errAsync != hipSuccess)
        cout << "Sync kernel error: " << hipGetErrorString(errAsync) << " in vp tree" << endl;
    #endif


    hipFree(d_tree);
    hipFree(d_dataset);
    hipFree(d_distances);
    hipFree(d_indexes);
    hipFree(d_vp_points);
}


void initIndexes(float *d, int n){
    dim3 block(512, 1);
    dim3 grid(CEIL_DIV(n, 512), 1);
    cudaInitIndexes<<<grid, block>>>(d, n);
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess){
        cout << "Sync kernel error: " << hipGetErrorString(errSync) << " in init indexing reduce: " << endl;
        exit(SYNCH_CUDA_ERROR);
    }
		
	if (errAsync != hipSuccess){
        cout << "Sync kernel error: " << hipGetErrorString(errSync) << " in init indexing: " << endl;
        exit(ASYNC_CUDA_ERROR);
    }
}


void copyIndexes(float *dest, float *source, int n){
    dim3 block(512, 1);
    dim3 grid(CEIL_DIV(n, 512), 1);
    cudaCopyIndexes<<<grid, block>>>(dest, source, n);
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess){
        cout << "Sync kernel error: " << hipGetErrorString(errSync) << " in init indexing reduce: " << endl;
        exit(SYNCH_CUDA_ERROR);
    }
		
	if (errAsync != hipSuccess){
        cout << "Sync kernel error: " << hipGetErrorString(errSync) << " in init indexing: " << endl;
        exit(ASYNC_CUDA_ERROR);
    }
}


__global__ void cudaInitIndexes(float *d, int n){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  
    if(thread_id < n){
        d[thread_id] = thread_id;
    }
}


__global__ void cudaCopyIndexes(float *dest, float *source, int n){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  
    if(thread_id < n){
        dest[thread_id] = source[thread_id];
    }
}
